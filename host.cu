#include "hip/hip_runtime.h"
/*  Host main routine   */
#include "headers.h"

using namespace cv;
using namespace std;

#define XIn 8
#define YIn 16

float* calcHistogram(float *magnitude, float*direction, int rows, int cols)
{

    hipError_t err = hipSuccess;

    size_t imageSize = rows*cols*sizeof(float);

    //final output array for feature vector of size 9*number of blocks
    float *final=(float*)malloc((9*imageSize/64));

    //creating the device array for the same
    float *d_hist_array=NULL;
    err = hipMalloc((void **)&d_hist_array,(9*imageSize/64));

    
    //creating the device array for magnitude
    float *d_magnitude = NULL;
    err = hipMalloc((void **)&d_magnitude, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for magnitude (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //copying the magnitude array into device magnitude array
    hipMemcpy(d_magnitude,magnitude, imageSize, hipMemcpyHostToDevice);
    
    
    //creating the device array for direction
    float *d_direction = NULL;
    err = hipMalloc((void **)&d_direction, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //copying the direction array into device direction array
    hipMemcpy(d_direction,direction, imageSize, hipMemcpyHostToDevice);
    
   
    //Specifying number of blocks and number of threads.
    dim3 grid(cols/8,rows/8,1);
    dim3 block(8,8,1);
    
    //calling the kernel
    histogram<<<grid,block>>>(d_magnitude,d_direction,rows,cols,d_hist_array);
   
    //copying the device array to host
    hipMemcpy(final, d_hist_array, (9*imageSize/64), hipMemcpyDeviceToHost);

    err = hipFree(d_hist_array);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_magnitude);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_direction);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Calculation of histogram done.\n");
    
    return(final);
}

int *typecastHistograms(float *histIn)
{
	int numElements = XIn*YIn*9;
    size_t size = numElements * sizeof(int);
	int *histOut = (int *)malloc(size);

	for(int i=0; i<numElements; ++i)
	{
		*(histOut + i) = (int)(*(histIn + i));
	}
	return histOut;
}

float *calcNormalHist(int *h_Hist, size_t sizeIn)
{
    bool DEBUG = false;
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Compute the size of normalized histogram
    int numElementsOut = (XIn - 1)*(YIn - 1)*36;
    size_t sizeOut = numElementsOut * sizeof(float);
    
    // Allocate the host output
    float *h_HistNorm = (float *)malloc(sizeOut);

    // Verify that allocations succeeded
    if (h_HistNorm == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate the device input
    int *d_Hist = NULL;
    err = hipMalloc((void **)&d_Hist, sizeIn);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output
    float *d_HistNorm = NULL;
    err = hipMalloc((void **)&d_HistNorm, sizeOut);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input in host memory to the device memory
    if(DEBUG) 
        printf("Copy input data from the host memory to the CUDA device\n");
    
    err = hipMemcpy(d_Hist, h_Hist, sizeIn, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the L2 Norm CUDA Kernel
    dim3 threadsPerBlock(XIn,YIn,1);
	dim3 blocksPerGrid(1,1,1);
	l2norm<<<blocksPerGrid, threadsPerBlock>>>(d_Hist, d_HistNorm);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch L2 Norm kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	hipDeviceSynchronize();

    // Copy the device result in device memory to the host memory
    if(DEBUG) 
        printf("Copy output data from the CUDA device to the host memory\n");
    
    err = hipMemcpy(h_HistNorm, d_HistNorm, sizeOut, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Free device memory
	err = hipFree(d_Hist);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_Hist (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_HistNorm);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_HistNorm (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// printf("Calculation of normalized histogram done.\n");
	
	// Return result
	return h_HistNorm;
}


int main(void)
{
    bool DEBUG = true;
    printf("Inside Host Code\n");
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    /*   Image Loading   */
    ifstream inFile("imageName.txt");
    ofstream outFile("outfileNeg.txt");

    int winsize = 3780;
    int blockSizeX = 18, blockSizeY = 2;
    int numBlocksPerWindowX = 7, numBlocksPerWindowY = 15;

    float bias;
    

    float *input;
    err = hipMalloc((void **)&input,winsize*sizeof(float));
    
    float *h_weights = (float *)malloc(winsize*sizeof(float));

    float *d_weights=NULL;
    err = hipMalloc((void **)&d_weights,(winsize*sizeof(float)));


    int opX = 1;
    int opY = 1;
    cout<<"opX: "<<opX<<" opY: "<<opY<<endl;
   
    float *h_svmScores = (float *)malloc(opX*opY*sizeof(float));

    float *d_svmScores = NULL;
    hipMalloc((void **)&d_svmScores,(opX*opY*sizeof(float)));

    FILE *f = fopen("svmweights.txt","r");
    for(int i = 0; i < winsize; i++)
        fscanf(f, "%f", h_weights+i);

    fscanf(f, "%f", &bias);
    fclose(f);

    hipMemcpy(d_weights, h_weights, winsize*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 grid1(opX,opY,1);
    dim3 block1(numBlocksPerWindowX*blockSizeX , 1 ,1);


    string line;
    while (getline(inFile, line))
    {
    

    // OpenCV code for reading image
    Mat origImg = imread(line,1);


    // To verify if original image is loaded properly 
    if(DEBUG)
    {        
        imshow("OriginalImage",origImg);
        waitKey(0);
    }  

    // resizing image
    int resolution = 1;
    int resizedRows  = origImg.rows/resolution;   
    int resizedCols  = origImg.cols/resolution;
    Mat resizeImg(resizedRows, resizedCols, CV_8UC3, Scalar(0,0,0));

    for (int k = 0; k < resizedRows; ++k)
    {
        for (int l = 0; l < resizedCols; ++l)
        {
            resizeImg.at<Vec3b>(k,l)[0] = origImg.at<Vec3b>(k*resolution,l*resolution)[0];
            resizeImg.at<Vec3b>(k,l)[1] = origImg.at<Vec3b>(k*resolution,l*resolution)[1];
            resizeImg.at<Vec3b>(k,l)[2] = origImg.at<Vec3b>(k*resolution,l*resolution)[2];

        }
    }   

    if(DEBUG)
    {    
        imshow("ResizedImage",resizeImg);
        waitKey(0);
    }
    
    int windowsRows = 128;
    int windowsCols = 64;

    // Step of each window
    int StepSlide = 32;

    for (int row = 0; row < resizeImg.rows - windowsRows; row += StepSlide)
        for (int col = 0; col < resizeImg.cols - windowsCols; col += StepSlide)
        {
           

            Mat img(windowsRows, windowsCols, CV_8UC3, Scalar(0,0,0));        
            for (int k = 0; k < windowsRows; ++k)
            {
                for (int l = 0; l < windowsCols; ++l)
                {
                    img.at<Vec3b>(k,l)[0] = resizeImg.at<Vec3b>(row+k,col+l)[0];
                    img.at<Vec3b>(k,l)[1] = resizeImg.at<Vec3b>(row+k,col+l)[1];
                    img.at<Vec3b>(k,l)[2] = resizeImg.at<Vec3b>(row+k,col+l)[2];
                }
            }

            // Padding required depending on kernel size
            // here kernel size is fixed always as 1 so 2*1
            int padding = 2;

            // Providing padding to image
            // X will be treated as rows and Y as cols
            int paddedX = img.rows + padding;
            int paddedY = img.cols + padding;

            size_t imageSize = img.rows * img.cols * sizeof(float);
            size_t paddedImageSize = paddedX * paddedY * sizeof(float);

            // Allocate memory for Blue Channel of image
            float *h_B = (float *)malloc(paddedImageSize);

            // Allocate memory for Green Channel of image
            float *h_G = (float *)malloc(paddedImageSize);

            // Allocate memory for Red Channel of image
            float *h_R = (float *)malloc(paddedImageSize);

            // Verify that allocations succeeded
            if (h_B == NULL || h_G == NULL || h_R == NULL)
            {
                fprintf(stderr, "Failed to allocate memory for image!\n");
                exit(EXIT_FAILURE);
            }

            // Converting Mat to 1D array 
            for (int i = 0; i < paddedX; ++i)
                for (int j = 0; j < paddedY; ++j)
                {
                    if( i==0 || i==paddedX-1 || j==0 || j==paddedY-1 )
                    {
                        h_B[i*paddedY + j] = 0;
                        h_G[i*paddedY + j] = 0;
                        h_R[i*paddedY + j] = 0;
                    }                
                    else
                    {
                        h_B[i*paddedY + j] = img.at<Vec3b>(i,j)[0];
                        h_G[i*paddedY + j] = img.at<Vec3b>(i,j)[1];
                        h_R[i*paddedY + j] = img.at<Vec3b>(i,j)[2];
                    }
                }


            // // Verify that the channel array is correct
            // if(DEBUG)
            // {   
            //     Mat checkImage(paddedX,paddedY, CV_8UC1, Scalar(0));
            //     for (int i = 0; i < paddedX*paddedY; ++i)
            //     {
            //         checkImage.at<uchar>(i/paddedY,i%paddedY) = h_B[i];
            //     }
            //     imshow("checkImage", checkImage);
            //     waitKey(0);
            // }

            
            // Allocate the device memory for Blue Channel
            float *d_B = NULL;
            err = hipMalloc((void **)&d_B, paddedImageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for B channel (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // Allocate the device memory for Green Channel
            float *d_G = NULL;
            err = hipMalloc((void **)&d_G, paddedImageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }


            // Allocate the device memory for Red Channel
            float *d_R = NULL;
            err = hipMalloc((void **)&d_R, paddedImageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            //  Copy image from host memory to device memory 
            if(DEBUG) 
                printf("Copying image from host memory to device memory.\n");
            
            err = hipMemcpy(d_B, h_B, paddedImageSize, hipMemcpyHostToDevice);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy blue channel image from host to device (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipMemcpy(d_G, h_G, paddedImageSize, hipMemcpyHostToDevice);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy green channel image from host to device (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipMemcpy(d_R, h_R, paddedImageSize, hipMemcpyHostToDevice);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy red channel image from host to device (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // Allocate memory for output gradient values: magnitude and angle
            float *h_outputMag = (float *)malloc(imageSize);
            float *h_outputAng = (float *)malloc(imageSize);

            // Allocate the device memory for output
            float *d_outputBMag = NULL;
            err = hipMalloc((void **)&d_outputBMag, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            float *d_outputBAng = NULL;
            err = hipMalloc((void **)&d_outputBAng, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // Allocate the device memory for output
            float *d_outputGMag = NULL;
            err = hipMalloc((void **)&d_outputGMag, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            float *d_outputGAng = NULL;
            err = hipMalloc((void **)&d_outputGAng, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // Allocate the device memory for output
            float *d_outputRMag = NULL;
            err = hipMalloc((void **)&d_outputRMag, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            float *d_outputRAng = NULL;
            err = hipMalloc((void **)&d_outputRAng, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // Allocate the device memory for output
            float *d_outputMag = NULL;
            err = hipMalloc((void **)&d_outputMag, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            float *d_outputAng = NULL;
            err = hipMalloc((void **)&d_outputAng, imageSize);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
            // Image is divided in no of image blocks gradients for each will be calculated parallely
            // Size of image block that will have its gradient calc. in one kernel call
            int blockX = 32, blockY = 32;
            dim3 X,Y;

            bool SHARED = true;  
            if(SHARED)
            {

                // Size to be allocated for shared memory inside kernel
                // This is the size of block along with padding so that convolution can be done 
                // at the border points
                int tileX = blockX + padding;
                int tileY = blockY + padding; 
                size_t tileSize = (tileX)*(tileY)*sizeof(float);

                // for each tile only 4th the threads are allocated and then reused accordingly
                int blockDimX = ceil((double)tileX/4), blockDimY = tileY;

                // the no. of thread blocks that have to be launched will be the no. of image rows and cols 
                // divided by the no. of pixel we wish to keep in one block of image
                int gridDimX = ceil((double)img.rows/blockX), gridDimY = ceil((double)img.cols/blockY);

                // Specifying execution configuration
                X.x = gridDimX, X.y = gridDimY;
                Y.x = blockDimX, Y.y = blockDimY;
                convolutionShared<<<X, Y, tileSize>>>(d_B, paddedX, paddedY, blockX, blockY, d_outputBMag, d_outputBAng, img.rows, img.cols);
                convolutionShared<<<X, Y, tileSize>>>(d_G, paddedX, paddedY, blockX, blockY, d_outputGMag, d_outputGAng, img.rows, img.cols);
                convolutionShared<<<X, Y, tileSize>>>(d_R, paddedX, paddedY, blockX, blockY, d_outputRMag, d_outputRAng, img.rows, img.cols);
           
            }else
            {
                // number of threads launched in X and Y axis
                Y.x = blockX, Y.y = blockY;

                // number of blocks launched in X and Y axis
                X.x = ceil(img.rows/blockX), X.y = ceil(img.cols/blockY);

                convolutionGlobal<<<X, Y>>>(d_B, paddedX, paddedY, d_outputBMag, d_outputBAng);
                convolutionGlobal<<<X, Y>>>(d_G, paddedX, paddedY, d_outputGMag, d_outputGAng);
                convolutionGlobal<<<X, Y>>>(d_R, paddedX, paddedY, d_outputRMag, d_outputRAng);

            }

            err = hipGetLastError();
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to launch convolution kernel (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
            hipDeviceSynchronize();

            Y.x = blockX, Y.y = blockY;
            max<<<X, Y>>>(d_outputBMag, d_outputBAng, d_outputGMag, d_outputGAng, d_outputRMag, d_outputRAng,
                d_outputMag, d_outputAng, img.rows, img.cols);
            
            // Copy the device result vector in device memory to the host result vector in host memory.
            if(DEBUG)     
                printf("Copy output data from the CUDA device to the host memory\n");

            err = hipMemcpy(h_outputMag, d_outputMag, imageSize, hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipMemcpy(h_outputAng, d_outputAng, imageSize, hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
            
            float* histGrad = calcHistogram(h_outputMag,h_outputAng,img.rows,img.cols);
                 
            
            // Verify that the resulting image is correct
            Mat magImage(img.rows, img.cols, CV_8UC1, Scalar(0));
            Mat angleImage(img.rows, img.cols, CV_8UC1, Scalar(0));
            for (int i = 0; i < img.rows*img.cols; ++i)
            {
                magImage.at<uchar>(i/img.cols,i%img.cols) = h_outputMag[i];
                angleImage.at<uchar>(i/img.cols,i%img.cols) = h_outputAng[i];
            }
            if(DEBUG)
            {   
                imshow("Output Angle", angleImage);
                imshow("Output Maginitude", magImage);
                waitKey(0);
            }
           

            // Free device global memory
            err = hipFree(d_outputBMag);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipFree(d_outputBAng);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipFree(d_B);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipFree(d_G);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to free device array G (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            err = hipFree(d_R);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to free device array R (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            
            // Free host memory
            free(h_B);
            free(h_G);
        	free(h_R);


        	// Size of input vector
        	int numElementsIn = XIn*YIn*9;
        	size_t sizeIn = numElementsIn * sizeof(int);
        	
        	// Calculate final feature vector from HOG
        	int *histOutput = typecastHistograms(histGrad);
        	float *featureVec = calcNormalHist(histOutput, sizeIn);

            int numElementsOut = (XIn - 1)*(YIn - 1)*36;
            
            if (outFile.is_open())
            {
                for(int count = 0; count < numElementsOut; count ++){
                    outFile << featureVec[count] << " " ;
                }
                outFile << endl;
            }
            else cout << "Unable to open file";

            // Verification of feature vector:
            // for (int i = 0; i < numElementsOut; ++i)
            // {
            //     cout<<"i: "<<i<<" featureVec:"<<featureVec[i]<<endl;
            // }

            hipMemcpy(input, featureVec, winsize*sizeof(float), hipMemcpyHostToDevice);

            LinearSVMEvaluation<<<grid1, block1>>>(input, h_weights, bias,
              blockSizeX, blockSizeY, numBlocksPerWindowX, numBlocksPerWindowY, d_svmScores);

            hipMemcpy(d_svmScores, h_svmScores, opX*opY*sizeof(float), hipMemcpyDeviceToHost);
            cout<<"Svm Score: "<<h_svmScores[0]<<endl;

            // depending on svm score we can classify this window as containing pedestrian or not
            if( h_svmScores[0]>0 )
            {
                Mat imgClone = resizeImg.clone();
                Rect r=Rect(col, row, windowsCols, windowsRows);
                rectangle(imgClone,r,Scalar(255,0,0),1,8,0);
                imshow("Pedestrian Detection", imgClone);
                waitKey(0);
            } 


            // Reset the device and exit
            // hipDeviceReset causes the driver to clean up all state. While
            // not mandatory in normal operation, it is good practice.  It is also
            // needed to ensure correct operation when the application is being
            // profiled. Calling hipDeviceReset causes all profile data to be
            // flushed before the application exits
            err = hipDeviceReset();

            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // printf("Done\n");
        }   

    }
    outFile.close(); 
    return 0;

}

